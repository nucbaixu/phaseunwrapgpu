
#include <hip/hip_runtime.h>

typedef  char sum_t[4];

///////////////////////////////////////////////////////////////
__device__ int  device_thresshold(float th, float v){
    
    if ((v <= th)&&( v>= -th))
        return 0;
    
    int mult =0;
    
    if      (v > th){
        while (v > mult*2*th){
            mult ++;
        }
        return mult;        
    }
    else if (v < th){
        
        while (v < mult*2*th){
            mult --;
        }
        return mult;
    }
}

///////////////////////////////////////////////////////////////
__device__ class Matrix{
private:
	float *_matrix;
	int _w,_h;

public:
	__device__
	Matrix (float *m, int w, int h): _matrix (m), _w(w), _h(h){}

	__device__
	float &getElem (int x, int y){  // the element to be retrieved depends on the block I am
		
		int voff = (blockIdx.x * 32) /_w;
		int hoff = blockIdx.x % (_w/32);

		int linesBefore = 32*voff;

		return _matrix[ (linesBefore*_w) + (32*hoff) + y*_w + x ];
	}

};

///////////////////////////////////////////////////////////////
__device__ void neigbourSum (Matrix &map, int levelWidth, sum_t *sums){

	int thid = threadIdx.x;
	int widthFrontier     = levelWidth/2;
	int RegionsPerSide    = 32/levelWidth;

	int target = thid/widthFrontier;
	int i = ((target*levelWidth) / 32) * levelWidth;            
	int j = (target*levelWidth) % 32;
	int e = thid%widthFrontier;

	int half = levelWidth/2;
	int f1   = half-1;
	int f2   = half;

	if ((i < 32) && (j < 32)){
		int sumsPerFrontier = levelWidth/2;	
		
		sums [thid * sumsPerFrontier + e][0] =  device_thresshold(0.5, map.getElem(i+e   , j+f1)   -  map.getElem(i+e   , j+f2));
		sums [thid * sumsPerFrontier + e][1] =  device_thresshold(0.5, map.getElem(i+f1  , j+f2+e) -  map.getElem(i+f2  , j+f2+e));
		sums [thid * sumsPerFrontier + e][2] =  device_thresshold(0.5, map.getElem(i+f2+e, j+f2)   -  map.getElem(i+f2+e, j+f1));
		sums [thid * sumsPerFrontier + e][3] =  device_thresshold(0.5, map.getElem(i+f2  , j+e)    -  map.getElem(i+f1  , j+e));
	}
}

///////////////////////////////////////////////////////////////
__device__ void computeTransform (sum_t *sums, int levelWidth){
	int thid = threadIdx.x;
	int sumsPerFrontier = levelWidth/2;	

	// reduce to one sum per region 
	if      (levelWidth == 2) { // 256 sums, 1 element each

		// done
	}
	else if (levelWidth == 4) { // 64 sums, 2 elements each
		if (threadIdx.x < 64*2){
			//int  threadI
		}
	}
	else if (levelWidth == 8) { // 16 sums, 4 elements each

	}
	else if (levelWidth == 16) { // 4 sums, 8 elements each

	}
	else if (levelWidth == 32) { // 1 sums, 16 elements	each

	}

	// ---  three levels inclusive scann  ---
	//scann the 4 elements of the sum to compute the transform
	if (levelWidth == 2) { // 256 sums

		// all 256 threads do work
		sums [thid][0] = sums [thid][0];
		sums [thid][0] = sums [thid][0] + sums [thid][1];
		sums [thid][0] = sums [thid][1] + sums [thid][2];
	}
	else if (levelWidth == 4) { // 64 sums
		
	}
	else if (levelWidth == 8) { // 16 sums

	}
	else if (levelWidth == 16) { // 4 sums

	}
	else if (levelWidth == 32) { // 1 sums

	}
}

///////////////////////////////////////////////////////////////
__device__ void regionCorrection(Matrix &map, sum_t *sums, int levelWidth){
	for (int pass =0; pass < 4; pass ++){
			int x = (blockDim.x*pass + threadIdx.x) / 32;
			int y = (blockDim.x*pass + threadIdx.x) % 32;

			// 4 regions, 
			int i = (x/(levelWidth/2));
			int j = (y/(levelWidth/2));
			
			int region = j*2 + i;

			i = (x/levelWidth);  // which column to be
			j = (y/levelWidth); // which row to be

			// TODO: aqui est� cascando, arregla estos indices, no estan bien del todo
		//	map.getElem(x,y) = sums[i + j*(32/levelWidth)][region];
	}
}

///////////////////////////////////////////////////////////////
/// this kernel operates in a Matrix, 
/// each block process a 32x32 matrix
/// solves 5 levels of the quad-tree aproach
__global__ void qt_unwrap_kernel (float *m, int w, int h, char *debug ){

	int levelWidth;
	Matrix map(m, w, h);
	__shared__ sum_t sum[256];

	for (levelWidth = 2; levelWidth <= 2; levelWidth*=2){

		// --- COMPUTE NEIGHBOUR SUM ---
		neigbourSum (map, levelWidth, sum);

		// --- SCAN SUMS ---
		computeTransform (sum, levelWidth);

		// --- APPLY TRANSFORMATION FOR EACH REGION ----
		// 256 threads need 4 passes to cover 32x32 matrix
		regionCorrection (map, sum, levelWidth);

		__syncthreads();
	}// tree levels

	
	// DEBUG THINGS: one block of 32x32
	debug[threadIdx.x*4]    = sum[threadIdx.x][0];
	debug[threadIdx.x*4 +1] = sum[threadIdx.x][1];
	debug[threadIdx.x*4 +2] = sum[threadIdx.x][2];
	debug[threadIdx.x*4 +3] = sum[threadIdx.x][3];
}