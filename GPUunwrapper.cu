#include "GPUunwrapper.h"

#include <hip/hip_runtime.h>
#include "QT_unwrap_kernel.h"

#include <iostream>

// local protorypes
void deviceEnumeration();


// class
GPUunwrapper::GPUunwrapper(float *m, int w, int h):h_map(m), _w(w), _h(h){}

GPUunwrapper::~GPUunwrapper(){
}

void GPUunwrapper::setMap(float *m, int w, int h){
	h_map = m;
	_w   = w;
	_h   = h;
}

float * GPUunwrapper::getMap(){
	return h_map;
}



bool GPUunwrapper::doUnwrap(){

	deviceEnumeration();

	float *d_map;
	size_t size = _w*_h*sizeof(float);

	if (hipMalloc (&d_map, size) != hipSuccess)
		throw -1;
	if (hipMemcpy (d_map, h_map, size, hipMemcpyHostToDevice) != hipSuccess)
		throw -2;
//	if (hipMemset (d_map, 0x0, size) != hipSuccess)
//		throw -3;

	char *d_debug;
	if (hipMalloc (&d_debug, 256*4*sizeof(char)) != hipSuccess)
		throw -4;
	if (hipMemset (d_debug, 0x0, 256*4*sizeof(char)) != hipSuccess)
		throw -5;

	int threadPerBlock = 256;
	int blocksPerGrid  = (_w*_h)/(32*32);

	qt_unwrap_kernel<<< blocksPerGrid, threadPerBlock >>> (d_map, _w, _h, d_debug);

 	if (hipMemcpy (h_map, d_map, size, hipMemcpyDeviceToHost) != hipSuccess)
		throw -3;

	char h_debug[256*4];
	if (hipMemcpy (h_debug, d_debug, 256*4*sizeof(char), hipMemcpyDeviceToHost) != hipSuccess)
		throw -3;

	// TODO: me he quedado aqui!
	std::cout << "SUMS: ";
	for (int i=0; i< 256*4; i+=4){
		std::cout << "[" << (int)h_debug[i] <<"," << (int)h_debug[i+1] <<"," << (int)h_debug[i+2] <<"," << (int)h_debug[i+3] << "]" << std::endl;
	}
	
	hipFree (d_map);

	return true;
}


/////////////////////////////////////////////////////////////////////////
// local functions

void deviceEnumeration(){

	int deviceCount =0;
	hipGetDeviceCount (&deviceCount);

	for (int i=0; i< deviceCount; i++){
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties (&deviceProp, i);
		if (deviceProp.major == 9999 && deviceProp.minor == 9999)
			printf ("not suporting CUDA\n");
		else{
			printf ("cuda device: %s\n", deviceProp.name);
		}	
	}
}
